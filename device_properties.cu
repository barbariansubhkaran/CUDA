#include <hip/hip_runtime.h>


#include <stdio.h>

void query_device()
{
   int deviceCount = 0;

   hipGetDeviceCount(&deviceCount);

   if(deviceCount == 0)
   {
       printf("No CUDA device found");
   }

   int devNo = 0;
   hipDeviceProp_t iProp;

   hipGetDeviceProperties(&iProp, devNo);

   printf("Device %d : %s\n", devNo, iProp.name);

   printf("Number of multiprocessors:               %d\n, iProp.multiprocessorCount");

   printf(" clock rate :                    %d\n, iProp.clockRate");

   printf(" Compute capability                 %d.%d\n", iProp.major, iProp.minor);


   printf(" Total amount of global memory :                  %4.2f KB\n", iProp.totalGlobalMem / 1024.0);

  printf(" Total amount of constant memory :               %4.2f  KB\n", iProp.sharedMemPerBlock / 1024.0);



}

int main()
{
       query_device();


  return 0;
}
