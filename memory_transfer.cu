#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void mem_trs_test(int * input)
{
     int gid = blockIdx.x * blockDim.x + threadIdx.x;

     printf("tid : %d, gid : %d, value : %d \n",threadIdx.x, gid, input[gid]);
}

int main()
{

   int size = 128;
   int byte_size = size * sizeof(int);

   int * h_input;

   h_input = (int*) malloc(byte_size);

   time t;

   srand((unsigned)time(&t));

   for(int i = 0; i < size; i++)
   {
      h_input[i] = (int)(rand() & 0xff);
   }


   int * d_input;

  hipMalloc((void**)&d_input,byte_size);

  hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);



   hipDeviceReset();
  return 0;
}
