#include <hip/hip_runtime.h>


#include <stdio.h>
#include <cstdlib>
#include <cassert>
#include <iostream>
#include <assert.h>
#include <math.h>
#include <stdlib.h>
#include <algorithm>
#include <vector>


using namespace std;


__global__ void vectorAdd(const int  *a,const int *b, int *c, int N)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;


    if(tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

void verify_result(vector<int> &a, vector<int> &b, vector<int> &c)
{
     for(int i = 0; i < a.size(); i++)
     {
         assert(c[i] == a[i] + b[i]);
     }
}


int main()
{
     constexpr int N = 1 << 16;

     constexpr size_t bytes =  sizeof(int) * N;


     vector<int> a;

     a.reserve(N);

     vector<int> b;

     b.reserve(N);

     vector<int> c;

     c.reserve(N);

     for(int i = 0; i < N; i++)
     {
         a.push_back(rand() % 100);
         b.push_back(rand() % 100);
     }
         int *d_a, *d_b, *d_c;

         hipMalloc(&d_a, bytes);
          hipMalloc(&d_b, bytes);
          hipMalloc(&d_c, bytes);



          hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
          hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);


          int NUM_THREADS = 1 << 10;

          int NUM_BLOCKS = (N + NUM_THREADS - 1 )/ NUM_THREADS;

         vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c,N);

         hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

         verify_result(a, b, c);

           hipFree(d_a);
           hipFree(d_b);
           hipFree(d_c);

           cout << "Completed " << endl;

           hipDeviceSynchronize();
           hipDeviceReset();




     return 0;
 }
