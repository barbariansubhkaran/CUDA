#include <hip/hip_runtime.h>


#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>


__global__ void occupancy_test(int * results)
{
     int gid = blockDim.x * blockIdx.x + threadIdx.x;

     int x1 = 1;
     int x2 = 2;
     int x3 = 3;
     int x4 = 4;
     int x5 = 5;
     int x6 = 6;
     int x7 = 7;
     int x8 = 8;

     results[gid] =  x1 + x2 + x3 + x4 + x5 + x6 + x7 + x8;

     

}
