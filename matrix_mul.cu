#include <hip/hip_runtime.h>


#include <stdio.h>
#include <cstdlib>
#include <cassert>
#include <iostream>


using namespace std;


__global__ void matrixMul(int *a, int *b, int *c, int N)
{
       int row = blockIdx.y * blockDim.y + threadIdx.y;
       int col = blockIdx.x * blockDim.x + threadIdx.x;

       if(row < N && col < N)
       {

            int tmp = 0;

            for(int i = 0; i < N; i++)
            {
                   tmp += a[row * N + i] * b[i * N + col];
            }

              c[row * N + col] = tmp;

       }


}

void init_matrix(int *m, int N)
{
    for(int i = 0; i < N * N; i++)
    {
        m[i] = rand() % 100;

    }
}

void verify_result(int *a, int *b, int *c, int N)
{
   int tmp;

   for(int i = 0; i < N; i++)
   {
       for(int j = 0; j < N; j++)
       {
            tmp = 0;
          for(int k = 0; k < N; k++)
          {
              tmp += a[i * N + k] * b[k * N +j];
          }

          assert(tmp == c[i * N +j]);
       }
   }
}

int main()
{

   int N = 1 << 10;

   size_t bytes = N * N * sizeof(int);

    int *a,  *b,  *c;

    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    init_matrix(a, N);
    init_matrix(b, N);


    int threads = 16;
    int blocks = (N + threads - 1) / threads;


    dim3 THREADS(threads,threads);
    dim3 BLOCKS(blocks, blocks);



 matrixMul<<<BLOCKS,THREADS >>>(a, b, c, N);


    hipDeviceSynchronize();
    hipDeviceReset();



    //verify_result(a, b, c, N);


  printf("Program success");



  return 0;
}
